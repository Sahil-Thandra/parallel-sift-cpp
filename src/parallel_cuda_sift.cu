#include "hip/hip_runtime.h"
#define _USE_MATH_DEFINES
#include <omp.h>
#include <chrono>
#include <cmath>
#include <iostream>
#include <vector>
#include <algorithm>
#include <array>
#include <tuple>
#include <cassert>

#include "parallel_cuda_sift.hpp"
#include "parallel_cuda_image.hpp"

using namespace std;
using namespace parallel_cuda_image;


namespace parallel_cuda_sift {

ScaleSpacePyramid generate_gaussian_pyramid(const Image& img, float sigma_min,
                                            int num_octaves, int scales_per_octave)
{
    // assume initial sigma is 1.0 (after resizing) and smooth
    // the image with sigma_diff to reach requried base_sigma
    float base_sigma = sigma_min / MIN_PIX_DIST;
    Image base_img = img.resize(img.width*2, img.height*2, Interpolation::BILINEAR);
    float sigma_diff = std::sqrt(base_sigma*base_sigma - 1.0f);
    base_img = gaussian_blur(base_img, sigma_diff);

    int imgs_per_octave = scales_per_octave + 3;

    // determine sigma values for bluring
    float k = std::pow(2, 1.0/scales_per_octave);
    std::vector<float> sigma_vals {base_sigma};
    for (int i = 1; i < imgs_per_octave; i++) {
        float sigma_prev = base_sigma * std::pow(k, i-1);
        float sigma_total = k * sigma_prev;
        sigma_vals.push_back(std::sqrt(sigma_total*sigma_total - sigma_prev*sigma_prev));
    }

    // create a scale space pyramid of gaussian images
    // images in each octave are half the size of images in the previous one
    ScaleSpacePyramid pyramid = {
        num_octaves,
        imgs_per_octave,
        std::vector<std::vector<Image>>(num_octaves)
    };
    
    // can't do parallelization here, since the current octave
    // depends on the previous octave (resize)
    for (int i = 0; i < num_octaves; i++) {
        pyramid.octaves[i].reserve(imgs_per_octave);
        pyramid.octaves[i].push_back(std::move(base_img));
        // can't do parallelization here, since the current image
        // depends on the previous image (gaussian blur)
        for (int j = 1; j < sigma_vals.size(); j++) {
            const Image& prev_img = pyramid.octaves[i].back();
            pyramid.octaves[i].push_back(gaussian_blur(prev_img, sigma_vals[j]));
        }
        // prepare base image for next octave
        const Image& next_base_img = pyramid.octaves[i][imgs_per_octave-3];
        base_img = next_base_img.resize(next_base_img.width/2, next_base_img.height/2,
                                        Interpolation::NEAREST);
    }
    return pyramid;
}

// generate pyramid of difference of gaussians (DoG) images
ScaleSpacePyramid generate_dog_pyramid(const ScaleSpacePyramid& img_pyramid)
{
    ScaleSpacePyramid dog_pyramid = {
        img_pyramid.num_octaves,
        img_pyramid.imgs_per_octave - 1,
        std::vector<std::vector<Image>>(img_pyramid.num_octaves)
    };
    // #pragma omp parallel for num_threads(8) 
    // increasing the processing time, cost of parallelization is high
    for (int i = 0; i < dog_pyramid.num_octaves; i++) {
        dog_pyramid.octaves[i].reserve(dog_pyramid.imgs_per_octave);
        // can't do parallelization here, since the current image
        // depends on the previous image
        for (int j = 1; j < img_pyramid.imgs_per_octave; j++) {
            Image diff = img_pyramid.octaves[i][j];
            // #pragma omp parallel for num_threads(16) 
            // increasing the processing time, cost of parallelization is high
            for (int pix_idx = 0; pix_idx < diff.size; pix_idx++) {
                diff.data[pix_idx] -= img_pyramid.octaves[i][j-1].data[pix_idx];
            }
            dog_pyramid.octaves[i].push_back(diff);
        }
    }

    return dog_pyramid;
}

/* bool point_is_extremum(const std::vector<Image>& octave, int scale, int x, int y)
{
    const Image& img = octave[scale];
    const Image& prev = octave[scale-1];
    const Image& next = octave[scale+1];

    bool is_min = true, is_max = true;
    float val = img.get_pixel(x, y, 0), neighbor;

    for (int dx : {-1,0,1}) {
        for (int dy : {-1,0,1}) {
            neighbor = prev.get_pixel(x+dx, y+dy, 0);
            if (neighbor > val) is_max = false;
            if (neighbor < val) is_min = false;

            neighbor = next.get_pixel(x+dx, y+dy, 0);
            if (neighbor > val) is_max = false;
            if (neighbor < val) is_min = false;

            neighbor = img.get_pixel(x+dx, y+dy, 0);
            if (neighbor > val) is_max = false;
            if (neighbor < val) is_min = false;

            if (!is_min && !is_max) return false;
        }
    }
    return true;
}
 */

__device__ float dev_get_pixel(const Image& img, int x, int y, int c) {
    if (x < 0)
        x = 0;
    if (x >= img.width)
        x = img.width - 1;
    if (y < 0)
        y = 0;
    if (y >= img.height)
        y = img.height - 1;
    return img.data[c*img.width*img.height + y*img.width + x];
}

__device__ bool point_is_extremum(const Image* octave, int scale, int x, int y)
{
    const Image& img = octave[scale];
    const Image& prev = octave[scale-1];
    const Image& next = octave[scale+1];

    bool is_min = true, is_max = true;
    float val = dev_get_pixel(img, x, y, 0), neighbor;

    for (int dx : {-1,0,1}) {
        for (int dy : {-1,0,1}) {
            neighbor = dev_get_pixel(prev, x+dx, y+dy, 0);
            if (neighbor > val) is_max = false;
            if (neighbor < val) is_min = false;

            neighbor = dev_get_pixel(next, x+dx, y+dy, 0);
            if (neighbor > val) is_max = false;
            if (neighbor < val) is_min = false;

            neighbor = dev_get_pixel(img, x+dx, y+dy, 0);
            if (neighbor > val) is_max = false;
            if (neighbor < val) is_min = false;

            if (!is_min && !is_max) return false;
        }
    }
    return true;
}


// fit a quadratic near the discrete extremum,
// update the keypoint (interpolated) extremum value
// and return offsets of the interpolated extremum from the discrete extremum

/* std::tuple<float, float, float> fit_quadratic(Keypoint& kp,
                                              const std::vector<Image>& octave,
                                              int scale)
{
    const Image& img = octave[scale];
    const Image& prev = octave[scale-1];
    const Image& next = octave[scale+1];

    float g1, g2, g3;
    float h11, h12, h13, h22, h23, h33;
    int x = kp.i, y = kp.j;

    // gradient 
    g1 = (next.get_pixel(x, y, 0) - prev.get_pixel(x, y, 0)) * 0.5;
    g2 = (img.get_pixel(x+1, y, 0) - img.get_pixel(x-1, y, 0)) * 0.5;
    g3 = (img.get_pixel(x, y+1, 0) - img.get_pixel(x, y-1, 0)) * 0.5;

    // hessian
    h11 = next.get_pixel(x, y, 0) + prev.get_pixel(x, y, 0) - 2*img.get_pixel(x, y, 0);
    h22 = img.get_pixel(x+1, y, 0) + img.get_pixel(x-1, y, 0) - 2*img.get_pixel(x, y, 0);
    h33 = img.get_pixel(x, y+1, 0) + img.get_pixel(x, y-1, 0) - 2*img.get_pixel(x, y, 0);
    h12 = (next.get_pixel(x+1, y, 0) - next.get_pixel(x-1, y, 0)
          -prev.get_pixel(x+1, y, 0) + prev.get_pixel(x-1, y, 0)) * 0.25;
    h13 = (next.get_pixel(x, y+1, 0) - next.get_pixel(x, y-1, 0)
          -prev.get_pixel(x, y+1, 0) + prev.get_pixel(x, y-1, 0)) * 0.25;
    h23 = (img.get_pixel(x+1, y+1, 0) - img.get_pixel(x+1, y-1, 0)
          -img.get_pixel(x-1, y+1, 0) + img.get_pixel(x-1, y-1, 0)) * 0.25;
    
    // invert hessian
    float hinv11, hinv12, hinv13, hinv22, hinv23, hinv33;
    float det = h11*h22*h33 - h11*h23*h23 - h12*h12*h33 + 2*h12*h13*h23 - h13*h13*h22;
    hinv11 = (h22*h33 - h23*h23) / det;
    hinv12 = (h13*h23 - h12*h33) / det;
    hinv13 = (h12*h23 - h13*h22) / det;
    hinv22 = (h11*h33 - h13*h13) / det;
    hinv23 = (h12*h13 - h11*h23) / det;
    hinv33 = (h11*h22 - h12*h12) / det;

    // find offsets of the interpolated extremum from the discrete extremum
    float offset_s = -hinv11*g1 - hinv12*g2 - hinv13*g3;
    float offset_x = -hinv12*g1 - hinv22*g2 - hinv23*g3;
    float offset_y = -hinv13*g1 - hinv23*g3 - hinv33*g3;

    float interpolated_extrema_val = img.get_pixel(x, y, 0)
                                   + 0.5*(g1*offset_s + g2*offset_x + g3*offset_y);
    kp.extremum_val = interpolated_extrema_val;
    return std::make_tuple(offset_s, offset_x, offset_y);
} */

__device__ Offsets fit_quadratic(Keypoint& kp, const Image* octave, int scale)
{
    Offsets kp_offsets;
    const Image& img = octave[scale];
    const Image& prev = octave[scale-1];
    const Image& next = octave[scale+1];

    float g1, g2, g3;
    float h11, h12, h13, h22, h23, h33;
    int x = kp.i, y = kp.j;

    // Gradient computation
    g1 = (dev_get_pixel(next, x, y, 0) - dev_get_pixel(prev, x, y, 0)) * 0.5f;
    g2 = (dev_get_pixel(img, x + 1, y, 0) - dev_get_pixel(img, x - 1, y, 0)) * 0.5f;
    g3 = (dev_get_pixel(img, x, y + 1, 0) - dev_get_pixel(img, x, y - 1, 0)) * 0.5f;

    // Hessian matrix computation
    h11 = dev_get_pixel(next, x, y, 0) + dev_get_pixel(prev, x, y, 0) - 2 * dev_get_pixel(img, x, y, 0);
    h22 = dev_get_pixel(img, x + 1, y, 0) + dev_get_pixel(img, x - 1, y, 0) - 2 * dev_get_pixel(img, x, y, 0);
    h33 = dev_get_pixel(img, x, y + 1, 0) + dev_get_pixel(img, x, y - 1, 0) - 2 * dev_get_pixel(img, x, y, 0);
    h12 = (dev_get_pixel(next, x + 1, y, 0) - dev_get_pixel(next, x - 1, y, 0)
          - dev_get_pixel(prev, x + 1, y, 0) + dev_get_pixel(prev, x - 1, y, 0)) * 0.25f;
    h13 = (dev_get_pixel(next, x, y + 1, 0) - dev_get_pixel(next, x, y - 1, 0)
          - dev_get_pixel(prev, x, y + 1, 0) + dev_get_pixel(prev, x, y - 1, 0)) * 0.25f;
    h23 = (dev_get_pixel(img, x + 1, y + 1, 0) - dev_get_pixel(img, x + 1, y - 1, 0)
          - dev_get_pixel(img, x - 1, y + 1, 0) + dev_get_pixel(img, x - 1, y - 1, 0)) * 0.25f;
    
    // Inverse Hessian computation
    float det = h11 * h22 * h33 - h11 * h23 * h23 - h12 * h12 * h33 + 2 * h12 * h13 * h23 - h13 * h13 * h22;
    float hinv11 = (h22 * h33 - h23 * h23) / det;
    float hinv12 = (h13 * h23 - h12 * h33) / det;
    float hinv13 = (h12 * h23 - h13 * h22) / det;
    float hinv22 = (h11 * h33 - h13 * h13) / det;
    float hinv23 = (h12 * h13 - h11 * h23) / det;
    float hinv33 = (h11 * h22 - h12 * h12) / det;

    // Calculate offset from the discrete extremum
    kp_offsets.s = -hinv11 * g1 - hinv12 * g2 - hinv13 * g3;
    kp_offsets.x = -hinv12 * g1 - hinv22 * g2 - hinv23 * g3;
    kp_offsets.y = -hinv13 * g1 - hinv23 * g2 - hinv33 * g3;

    float interpolated_extrema_val = dev_get_pixel(img, x, y, 0)
                                   + 0.5f * (g1 * kp_offsets.s + g2 * kp_offsets.x+ g3 * kp_offsets.y);
    kp.extremum_val = interpolated_extrema_val;

    return kp_offsets;
}

/* bool point_is_on_edge(const Keypoint& kp, const std::vector<Image>& octave, float edge_thresh=C_EDGE)
{
    const Image& img = octave[kp.scale];
    float h11, h12, h22;
    int x = kp.i, y = kp.j;
    h11 = img.get_pixel(x+1, y, 0) + img.get_pixel(x-1, y, 0) - 2*img.get_pixel(x, y, 0);
    h22 = img.get_pixel(x, y+1, 0) + img.get_pixel(x, y-1, 0) - 2*img.get_pixel(x, y, 0);
    h12 = (img.get_pixel(x+1, y+1, 0) - img.get_pixel(x+1, y-1, 0)
          -img.get_pixel(x-1, y+1, 0) + img.get_pixel(x-1, y-1, 0)) * 0.25;

    float det_hessian = h11*h22 - h12*h12;
    float tr_hessian = h11 + h22;
    float edgeness = tr_hessian*tr_hessian / det_hessian;
    if (edgeness > std::pow(edge_thresh+1, 2)/edge_thresh)
        return true;
    else
        return false;
} */

__device__ bool point_is_on_edge(const Keypoint& kp, const Image* octave, float edge_thresh = C_EDGE)
{
    const Image& img = octave[kp.scale];
    float h11, h12, h22;
    int x = kp.i, y = kp.j;

    // Second derivative computation using device-specific pixel access
    h11 = dev_get_pixel(img, x+1, y, 0) + dev_get_pixel(img, x-1, y, 0) - 2 * dev_get_pixel(img, x, y, 0);
    h22 = dev_get_pixel(img, x, y+1, 0) + dev_get_pixel(img, x, y-1, 0) - 2 * dev_get_pixel(img, x, y, 0);
    h12 = (dev_get_pixel(img, x+1, y+1, 0) - dev_get_pixel(img, x+1, y-1, 0)
          - dev_get_pixel(img, x-1, y+1, 0) + dev_get_pixel(img, x-1, y-1, 0)) * 0.25;

    // Hessian determinant and trace calculation for edgeness check
    float det_hessian = h11 * h22 - h12 * h12;
    float tr_hessian = h11 + h22;
    float edgeness = tr_hessian * tr_hessian / det_hessian;

    // Edge response check against threshold
    return edgeness > (pow(edge_thresh + 1, 2) / edge_thresh);
}

/* void find_input_img_coords(Keypoint& kp, float offset_s, float offset_x, float offset_y,
                                   float sigma_min=SIGMA_MIN,
                                   float min_pix_dist=MIN_PIX_DIST, int n_spo=N_SPO)
{
    kp.sigma = std::pow(2, kp.octave) * sigma_min * std::pow(2, (offset_s+kp.scale)/n_spo);
    kp.x = min_pix_dist * std::pow(2, kp.octave) * (offset_x+kp.i);
    kp.y = min_pix_dist * std::pow(2, kp.octave) * (offset_y+kp.j);
} */

__device__ void find_input_img_coords(Keypoint& kp, float offset_s, float offset_x, float offset_y,
                                      float sigma_min = SIGMA_MIN,
                                      float min_pix_dist = MIN_PIX_DIST, int n_spo = N_SPO)
{
    kp.sigma = powf(2, kp.octave) * sigma_min * powf(2, (offset_s + kp.scale) / n_spo);
    kp.x = min_pix_dist * powf(2, kp.octave) * (offset_x + kp.i);
    kp.y = min_pix_dist * powf(2, kp.octave) * (offset_y + kp.j);
}

/* bool refine_or_discard_keypoint(Keypoint& kp, const std::vector<Image>& octave,
                                 float contrast_thresh, float edge_thresh)
{
    int k = 0;
    bool kp_is_valid = false; 
    while (k++ < MAX_REFINEMENT_ITERS) {
        std::tuple<float, float, float> result = fit_quadratic(kp, octave, kp.scale);
        float offset_s = std::get<0>(result);
        float offset_x = std::get<1>(result);
        float offset_y = std::get<2>(result);

        float max_offset = std::max({std::abs(offset_s),
                                     std::abs(offset_x),
                                     std::abs(offset_y)});
        // find nearest discrete coordinates
        kp.scale += std::round(offset_s);
        kp.i += std::round(offset_x);
        kp.j += std::round(offset_y);
        if (kp.scale >= octave.size()-1 || kp.scale < 1)
            break;

        bool valid_contrast = std::abs(kp.extremum_val) > contrast_thresh;
        if (max_offset < 0.6 && valid_contrast && !point_is_on_edge(kp, octave, edge_thresh)) {
            find_input_img_coords(kp, offset_s, offset_x, offset_y);
            kp_is_valid = true;
            break;
        }
    }
    return kp_is_valid;
} */

__device__ float device_abs(float x) {
    return x < 0 ? -x : x;
}

__device__ float device_max(float a, float b, float c) {
    return fmaxf(a, fmaxf(b, c));
}

__device__ bool refine_or_discard_keypoint(Keypoint& kp, const Image* octave, int num_octaves,
                                           float contrast_thresh, float edge_thresh)
{
    int k = 0;
    bool kp_is_valid = false;
    Offsets kp_offsets;
    while (k++ < MAX_REFINEMENT_ITERS) {
        kp_offsets = fit_quadratic(kp, octave, kp.scale);

        float max_offset = device_max(device_abs(kp_offsets.s), device_abs(kp_offsets.x), device_abs(kp_offsets.y));

        // Update kp values
        int new_scale = kp.scale + roundf(kp_offsets.s);
        int new_i = kp.i + roundf(kp_offsets.x);
        int new_j = kp.j + roundf(kp_offsets.y);
        
        if (new_scale >= num_octaves - 1 || new_scale < 1)
            break;

        kp.scale = new_scale;
        kp.i = new_i;
        kp.j = new_j;

        if (device_abs(kp.extremum_val) > contrast_thresh && max_offset < 0.6 && !point_is_on_edge(kp, octave, edge_thresh)) {
            find_input_img_coords(kp, kp_offsets.s, kp_offsets.x, kp_offsets.y);
            kp_is_valid = true;
            break;
        }
    }
    return kp_is_valid;
}

/* std::vector<Keypoint> find_keypoints(const ScaleSpacePyramid& dog_pyramid, float contrast_thresh,
                                     float edge_thresh)
{
    std::vector<Keypoint> keypoints;
    // #pragma omp parallel for num_threads(16) 
    // increasing the processing time, cost of parallelization is high
    for (int i = 0; i < dog_pyramid.num_octaves; i++) {
        const std::vector<Image>& octave = dog_pyramid.octaves[i];
        // #pragma omp parallel for num_threads(16)
        for (int j = 1; j < dog_pyramid.imgs_per_octave-1; j++) {
            const Image& img = octave[j];
            #pragma omp parallel for collapse(2) num_threads(16)
            for (int x = 1; x < img.width-1; x++) {
                for (int y = 1; y < img.height-1; y++) {
                    if (std::abs(img.get_pixel(x, y, 0)) < 0.8*contrast_thresh) {
                        continue;
                    }
                    if (point_is_extremum(octave, j, x, y)) {
                        Keypoint kp = {x, y, i, j, -1, -1, -1, -1};
                        bool kp_is_valid = refine_or_discard_keypoint(kp, octave, contrast_thresh,
                                                                      edge_thresh);
                        if (kp_is_valid) {
                            #pragma omp critical 
                            {
                                keypoints.push_back(kp);
                            }
                        }
                    }
                }
            }
        }
    }
    return keypoints;
} */

__global__ void detectKeypoints(Image* octave, int octaveIndex, int imgIndex, int imgsPerOctave, float contrastThresh, float edgeThresh, Keypoint* keypoints, int* keypointCount, int maxKeypoints) {

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x > 0 && y > 0 && x < octave[imgIndex].width - 1 && y < octave[imgIndex].height - 1) {
        float pixelValue = dev_get_pixel(octave[imgIndex], x, y, 0);
        if (fabs(pixelValue) < 0.8 * contrastThresh) {
            return;
        }

        if (point_is_extremum(octave, imgIndex, x, y)) {
            Keypoint kp = {x, y, octaveIndex, imgIndex, -1, -1, -1, -1};
            if (refine_or_discard_keypoint(kp, octave, imgsPerOctave, contrastThresh, edgeThresh)) {
                int index = atomicAdd(keypointCount, 1);
                if (index < maxKeypoints) {
                    keypoints[index] = kp;
                }
            }
        }
    }
}

__global__ void imageDataRefCopy(Image* dev_octave, const int img_index, float *dev_img)
{
    if (threadIdx.x == 0 && blockIdx.x == 0)
    {
        dev_octave[img_index].data = dev_img;
    }
}

__global__ void setDeviceImageSize(Image* dev_octave, const int img_index, const int w, const int h, const int c){
    if (threadIdx.x == 0 && blockIdx.x == 0)
    {
        dev_octave[img_index].width = w;
        dev_octave[img_index].height = h;
        dev_octave[img_index].channels = c;
        dev_octave[img_index].size = w*h*c;
    }
}

std::vector<Keypoint> find_keypoints(const ScaleSpacePyramid& dog_pyramid, float contrast_thresh,
                                     float edge_thresh)
{
    int* dev_kp_count;
    hipMalloc(&dev_kp_count, sizeof(int));
    hipMemset(dev_kp_count, 0, sizeof(int));

    Keypoint* dev_keypoints;
    int maxKeypoints = 10000;
    hipMalloc(&dev_keypoints, sizeof(Keypoint) * maxKeypoints);

    std::vector<Keypoint> keypoints;

    for (int i = 0; i < dog_pyramid.num_octaves; i++) 
    {
        const std::vector<Image>& octave = dog_pyramid.octaves[i];

        Image* dev_octave_img[dog_pyramid.imgs_per_octave];
        float *dev_octave_img_data[dog_pyramid.imgs_per_octave];
        hipError_t err;

        err = hipMalloc((void**)&dev_octave_img, dog_pyramid.imgs_per_octave * sizeof(Image));
        if (err != hipSuccess){
            std::cout<<hipGetErrorString(err)<<std::endl;
            exit(-1);
        }

        // copy images in an octave to GPU
        for(int imgOctIndex = 0; imgOctIndex< dog_pyramid.imgs_per_octave; imgOctIndex++)
        {

            setDeviceImageSize<<<1,1>>>(*dev_octave_img, imgOctIndex, octave[imgOctIndex].width, octave[imgOctIndex].height, octave[imgOctIndex].channels);
            hipDeviceSynchronize();

            err = hipMalloc((void**)&dev_octave_img_data[imgOctIndex], octave[imgOctIndex].size * sizeof(float));
            if (err != hipSuccess){
                std::cout<<hipGetErrorString(err)<<std::endl;
                exit(-1);
            }

            hipMemcpy(dev_octave_img_data[imgOctIndex], octave[imgOctIndex].data, octave[imgOctIndex].size*sizeof(float), hipMemcpyHostToDevice);
            imageDataRefCopy<<<1,1>>>(*dev_octave_img, imgOctIndex, dev_octave_img_data[imgOctIndex]);
            hipDeviceSynchronize();
        }
        for (int j = 1; j < dog_pyramid.imgs_per_octave-1; j++) {
            const Image& img = octave[j];
            dim3 threadsPerBlock(16, 16);
            dim3 numBlocks((img.width + threadsPerBlock.x - 1) / threadsPerBlock.x, 
                    (img.height + threadsPerBlock.y - 1) / threadsPerBlock.y);

            detectKeypoints<<<numBlocks, threadsPerBlock>>>(*dev_octave_img, i, j, dog_pyramid.imgs_per_octave, contrast_thresh, edge_thresh, dev_keypoints, dev_kp_count, maxKeypoints);
            hipDeviceSynchronize();
        }
        // freeing GPU space    
        for(int imgOctIndex = 0; imgOctIndex< dog_pyramid.imgs_per_octave; imgOctIndex++) {
            hipFree(dev_octave_img_data[imgOctIndex]);
        }
        hipFree(dev_octave_img);

    }
    
    int keyPointCount = 0;

    hipMemcpy(&keyPointCount, dev_kp_count, sizeof(int), hipMemcpyDeviceToHost);
    keypoints.resize(keyPointCount);

    hipError_t cudaStatus = hipMemcpy(keypoints.data(), dev_keypoints, sizeof(Keypoint) * keyPointCount, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipMemcpy failed!" << std::endl;
    }

    hipFree(dev_keypoints);
    hipFree(dev_kp_count);

    return keypoints;
}

// calculate x and y derivatives for all images in the input pyramid
ScaleSpacePyramid generate_gradient_pyramid(const ScaleSpacePyramid& pyramid)
{
    ScaleSpacePyramid grad_pyramid = {
        pyramid.num_octaves,
        pyramid.imgs_per_octave,
        std::vector<std::vector<Image>>(pyramid.num_octaves)
    };
    // #pragma omp parallel for num_threads(16) 
    // increasing the processing time, cost of parallelization is high
    for (int i = 0; i < pyramid.num_octaves; i++) {
        grad_pyramid.octaves[i].reserve(grad_pyramid.imgs_per_octave);
        int width = pyramid.octaves[i][0].width;
        int height = pyramid.octaves[i][0].height;
        // #pragma omp parallel for num_threads(16)
        for (int j = 0; j < pyramid.imgs_per_octave; j++) {
            Image grad(width, height, 2);
            float gx, gy;
            // #pragma omp parallel for collapse(2) num_threads(16) 
            // increasing the processing time, cost of parallelization is high
            for (int x = 1; x < grad.width-1; x++) {
                for (int y = 1; y < grad.height-1; y++) {
                    gx = (pyramid.octaves[i][j].get_pixel(x+1, y, 0)
                         -pyramid.octaves[i][j].get_pixel(x-1, y, 0)) * 0.5;
                    grad.set_pixel(x, y, 0, gx);
                    gy = (pyramid.octaves[i][j].get_pixel(x, y+1, 0)
                         -pyramid.octaves[i][j].get_pixel(x, y-1, 0)) * 0.5;
                    grad.set_pixel(x, y, 1, gy);
                }
            }
            // #pragma omp critical
            {
                grad_pyramid.octaves[i].push_back(grad);
            }
        }
    }
    return grad_pyramid;
}

// convolve 6x with box filter
void smooth_histogram(float hist[N_BINS])
{
    float tmp_hist[N_BINS];
    // can't do parallelization here, small number of iterations
    for (int i = 0; i < 6; i++) {
        for (int j = 0; j < N_BINS; j++) {
            int prev_idx = (j-1+N_BINS)%N_BINS;
            int next_idx = (j+1)%N_BINS;
            tmp_hist[j] = (hist[prev_idx] + hist[j] + hist[next_idx]) / 3;
        }
        for (int j = 0; j < N_BINS; j++) {
            hist[j] = tmp_hist[j];
        }
    }
}

std::vector<float> find_keypoint_orientations(Keypoint& kp, 
                                              const ScaleSpacePyramid& grad_pyramid,
                                              float lambda_ori, float lambda_desc)
{
    float pix_dist = MIN_PIX_DIST * std::pow(2, kp.octave);
    const Image& img_grad = grad_pyramid.octaves[kp.octave][kp.scale];

    // discard kp if too close to image borders 
    float min_dist_from_border = std::min({kp.x, kp.y, pix_dist*img_grad.width-kp.x,
                                           pix_dist*img_grad.height-kp.y});
    if (min_dist_from_border <= std::sqrt(2)*lambda_desc*kp.sigma) {
        return {};
    }

    float hist[N_BINS] = {};
    int bin;
    float gx, gy, grad_norm, weight, theta;
    float patch_sigma = lambda_ori * kp.sigma;
    float patch_radius = 3 * patch_sigma;
    int x_start = std::round((kp.x - patch_radius)/pix_dist);
    int x_end = std::round((kp.x + patch_radius)/pix_dist);
    int y_start = std::round((kp.y - patch_radius)/pix_dist);
    int y_end = std::round((kp.y + patch_radius)/pix_dist);

    // accumulate gradients in orientation histogram
    
    // #pragma omp parallel for collapse(2) num_threads(16) 
    // increasing the processing time due to critical region
    for (int x = x_start; x <= x_end; x++) {
        for (int y = y_start; y <= y_end; y++) {
            gx = img_grad.get_pixel(x, y, 0);
            gy = img_grad.get_pixel(x, y, 1);
            grad_norm = std::sqrt(gx*gx + gy*gy);
            weight = std::exp(-(std::pow(x*pix_dist-kp.x, 2)+std::pow(y*pix_dist-kp.y, 2))
                              /(2*patch_sigma*patch_sigma));
            theta = std::fmod(std::atan2(gy, gx)+2*M_PI, 2*M_PI);
            bin = (int)std::round(N_BINS/(2*M_PI)*theta) % N_BINS;
            // #pragma omp critical 
            {
                hist[bin] += weight * grad_norm;
            }
        }
    }

    smooth_histogram(hist);

    // extract reference orientations
    float ori_thresh = 0.8, ori_max = 0;
    std::vector<float> orientations;
    // can't do parallelization here, small number of iterations
    for (int j = 0; j < N_BINS; j++) {
        if (hist[j] > ori_max) {
            ori_max = hist[j];
        }
    }
    // can't do parallelization here, small number of iterations
    for (int j = 0; j < N_BINS; j++) {
        if (hist[j] >= ori_thresh * ori_max) {
            float prev = hist[(j-1+N_BINS)%N_BINS], next = hist[(j+1)%N_BINS];
            if (prev > hist[j] || next > hist[j])
                continue;
            float theta = 2*M_PI*(j+1)/N_BINS + M_PI/N_BINS*(prev-next)/(prev-2*hist[j]+next);
            orientations.push_back(theta);
        }
    }
    return orientations;
}

void update_histograms(float hist[N_HIST][N_HIST][N_ORI], float x, float y,
                       float contrib, float theta_mn, float lambda_desc)
{
    float x_i, y_j;
    // can't do parallelization here, small number of iterations
    for (int i = 1; i <= N_HIST; i++) {
        x_i = (i-(1+(float)N_HIST)/2) * 2*lambda_desc/N_HIST;
        if (std::abs(x_i-x) > 2*lambda_desc/N_HIST)
            continue;
        for (int j = 1; j <= N_HIST; j++) {
            y_j = (j-(1+(float)N_HIST)/2) * 2*lambda_desc/N_HIST;
            if (std::abs(y_j-y) > 2*lambda_desc/N_HIST)
                continue;
            
            float hist_weight = (1 - N_HIST*0.5/lambda_desc*std::abs(x_i-x))
                               *(1 - N_HIST*0.5/lambda_desc*std::abs(y_j-y));

            for (int k = 1; k <= N_ORI; k++) {
                float theta_k = 2*M_PI*(k-1)/N_ORI;
                float theta_diff = std::fmod(theta_k-theta_mn+2*M_PI, 2*M_PI);
                if (std::abs(theta_diff) >= 2*M_PI/N_ORI)
                    continue;
                float bin_weight = 1 - N_ORI*0.5/M_PI*std::abs(theta_diff);
                hist[i-1][j-1][k-1] += hist_weight*bin_weight*contrib;
            }
        }
    }
}

void hists_to_vec(float histograms[N_HIST][N_HIST][N_ORI], std::array<uint8_t, 128>& feature_vec)
{
    int size = N_HIST*N_HIST*N_ORI;
    float *hist = reinterpret_cast<float *>(histograms);

    float norm = 0;
    for (int i = 0; i < size; i++) {
        norm += hist[i] * hist[i];
    }
    norm = std::sqrt(norm);
    float norm2 = 0;
    for (int i = 0; i < size; i++) {
        hist[i] = std::min(hist[i], 0.2f*norm);
        norm2 += hist[i] * hist[i];
    }
    norm2 = std::sqrt(norm2);
    for (int i = 0; i < size; i++) {
        float val = std::floor(512*hist[i]/norm2);
        feature_vec[i] = std::min((int)val, 255);
    }
}

void compute_keypoint_descriptor(Keypoint& kp, float theta,
                                 const ScaleSpacePyramid& grad_pyramid,
                                 float lambda_desc)
{
    float pix_dist = MIN_PIX_DIST * std::pow(2, kp.octave);
    const Image& img_grad = grad_pyramid.octaves[kp.octave][kp.scale];
    float histograms[N_HIST][N_HIST][N_ORI] = {{{0}}};

    //find start and end coords for loops over image patch
    float half_size = std::sqrt(2)*lambda_desc*kp.sigma*(N_HIST+1.)/N_HIST;
    int x_start = std::round((kp.x-half_size) / pix_dist);
    int x_end = std::round((kp.x+half_size) / pix_dist);
    int y_start = std::round((kp.y-half_size) / pix_dist);
    int y_end = std::round((kp.y+half_size) / pix_dist);

    float cos_t = std::cos(theta), sin_t = std::sin(theta);
    float patch_sigma = lambda_desc * kp.sigma;
    //accumulate samples into histograms
    #pragma omp parallel for collapse(2) num_threads(16)
    for (int m = x_start; m <= x_end; m++) {
        for (int n = y_start; n <= y_end; n++) {
            // find normalized coords w.r.t. kp position and reference orientation
            float x = ((m*pix_dist - kp.x)*cos_t
                      +(n*pix_dist - kp.y)*sin_t) / kp.sigma;
            float y = (-(m*pix_dist - kp.x)*sin_t
                       +(n*pix_dist - kp.y)*cos_t) / kp.sigma;

            // verify (x, y) is inside the description patch
            if (std::max(std::abs(x), std::abs(y)) > lambda_desc*(N_HIST+1.)/N_HIST)
                continue;

            float gx = img_grad.get_pixel(m, n, 0), gy = img_grad.get_pixel(m, n, 1);
            float theta_mn = std::fmod(std::atan2(gy, gx)-theta+4*M_PI, 2*M_PI);
            float grad_norm = std::sqrt(gx*gx + gy*gy);
            float weight = std::exp(-(std::pow(m*pix_dist-kp.x, 2)+std::pow(n*pix_dist-kp.y, 2))
                                    /(2*patch_sigma*patch_sigma));
            float contribution = weight * grad_norm;

            update_histograms(histograms, x, y, contribution, theta_mn, lambda_desc);
        }
    }

    // build feature vector (descriptor) from histograms
    hists_to_vec(histograms, kp.descriptor);
}


std::vector<Keypoint> find_keypoints_and_descriptors(const Image& img, float sigma_min,
                                                     int num_octaves, int scales_per_octave, 
                                                     float contrast_thresh, float edge_thresh, 
                                                     float lambda_ori, float lambda_desc)
{
    assert(img.channels == 1 || img.channels == 3);

    const Image& input = img.channels == 1 ? img : rgb_to_grayscale(img);
    auto start = std::chrono::high_resolution_clock::now();
    ScaleSpacePyramid gaussian_pyramid = generate_gaussian_pyramid(input, sigma_min, num_octaves,
                                                                   scales_per_octave);
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed = end - start;
    std::cout << "Time to generate gaussian pyramid: " << elapsed.count() << "s" << std::endl;

    start = std::chrono::high_resolution_clock::now();
    ScaleSpacePyramid dog_pyramid = generate_dog_pyramid(gaussian_pyramid);
    end = std::chrono::high_resolution_clock::now();
    elapsed = end - start;
    std::cout << "Time to generate difference of gaussian pyramid: " << elapsed.count() << "s" << std::endl;

    start = std::chrono::high_resolution_clock::now();
    std::vector<Keypoint> tmp_kps = find_keypoints(dog_pyramid, contrast_thresh, edge_thresh);
    end = std::chrono::high_resolution_clock::now();
    elapsed = end - start;
    std::cout << "Time to find valid keypoints: " << elapsed.count() << "s" << std::endl;

    start = std::chrono::high_resolution_clock::now();
    ScaleSpacePyramid grad_pyramid = generate_gradient_pyramid(gaussian_pyramid);
    end = std::chrono::high_resolution_clock::now();
    elapsed = end - start;
    std::cout << "Time to generate gradient pyramid: " << elapsed.count() << "s" << std::endl;
    
    std::vector<Keypoint> kps;

    start = std::chrono::high_resolution_clock::now();
    #pragma omp parallel for num_threads(16)
    for (int i = 0; i< tmp_kps.size(); i++) {
        std::vector<float> orientations = find_keypoint_orientations(tmp_kps[i], grad_pyramid,
                                                                     lambda_ori, lambda_desc);
        for (float theta : orientations) {
            Keypoint kp = tmp_kps[i];
            compute_keypoint_descriptor(kp, theta, grad_pyramid, lambda_desc);
            #pragma omp critical
            {
                kps.push_back(kp);
            }
        }
    }
    end = std::chrono::high_resolution_clock::now();
    elapsed = end - start;
    std::cout << "Time to find key points orientation and compute descriptor: " << elapsed.count() << "s" << std::endl;

    return kps;
}

// float euclidean_dist(std::array<uint8_t, 128>& a, std::array<uint8_t, 128>& b)
// {
//     float dist = 0;
//     for (int i = 0; i < 128; i++) {
//         int di = (int)a[i] - b[i];
//         dist += di * di;
//     }
//     return std::sqrt(dist);
// }

__device__ float euclidean_dist(uint8_t* a, uint8_t* b)
{
    float dist = 0;
    for (int i = 0; i < 128; i++) {
        int di = (int)a[i] - b[i];
        dist += di * di;
    }
    return sqrt(dist);
}

__global__ void find_matches(uint8_t* a_descriptors, uint8_t* b_descriptors, int* matches, int a_size, int b_size, int desc_length, float thresh_relative, float thresh_absolute) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < a_size) {
        int nn1_idx = -1;
        float nn1_dist = 100000000, nn2_dist = 100000000;
        for (int j = 0; j < b_size; j++) {
            float dist = euclidean_dist(&a_descriptors[i * desc_length], &b_descriptors[j * desc_length]);
            if (dist < nn1_dist) {
                nn2_dist = nn1_dist;
                nn1_dist = dist;
                nn1_idx = j;
            } else if (nn1_dist <= dist && dist < nn2_dist) {
                nn2_dist = dist;
            }
        }
        if (nn1_dist < thresh_relative * nn2_dist && nn1_dist < thresh_absolute) {
            int idx = atomicAdd(matches, 1);
            // printf("matches[0]=%d\n", *matches);
            // printf("idx=%d, match %d,%d\n", idx, i, nn1_idx);
            matches[2 * idx + 1] = i;
            matches[2 * idx + 2] = nn1_idx;
        }
    }
}
// std::vector<std::pair<int, int>> find_keypoint_matches(std::vector<Keypoint>& a,
//                                                        std::vector<Keypoint>& b,
//                                                        float thresh_relative,
//                                                        float thresh_absolute)
// {
//     assert(a.size() >= 2 && b.size() >= 2);

//     std::vector<std::pair<int, int>> matches;

//     #pragma omp parallel for num_threads(16)
//     for (int i = 0; i < a.size(); i++) {
//         // find two nearest neighbours in b for current keypoint from a
//         int nn1_idx = -1;
//         float nn1_dist = 100000000, nn2_dist = 100000000;
//         // can't do parallelization here, because we are trying to 
//         // find minimum distance across iterations
//         for (int j = 0; j < b.size(); j++) {
//             float dist = euclidean_dist(a[i].descriptor, b[j].descriptor);
//             if (dist < nn1_dist) {
//                 nn2_dist = nn1_dist;
//                 nn1_dist = dist;
//                 nn1_idx = j;
//             } else if (nn1_dist <= dist && dist < nn2_dist) {
//                 nn2_dist = dist;
//             }
//         }
//         if (nn1_dist < thresh_relative*nn2_dist && nn1_dist < thresh_absolute) {
//             #pragma omp critical
//             {
//                 matches.push_back({i, nn1_idx});
//             }
//         }
//     }
//     return matches;
// }

__global__ void imageDesc(uint8_t* d_a_descriptors, int size)
{
    
    if (threadIdx.x == 0 && blockIdx.x == 0)
    {
        printf("Here at CUDA dec print\n");
        for (int k = 0; k < size; k++) {
            if((k<10) || k>(size-10))
            {
                printf("%d\n", d_a_descriptors[k*128]);
            }       
        }
    } 
}
__global__ void matchPrint(int* match)
{
    printf("Here at CUDA match print\n");
    if (threadIdx.x == 0 && blockIdx.x == 0)
    {
        for (int k = 0; k < match[0]; k++) {
                printf("aID=%d , bID=%d\n", match[k], match[k+1]);  
        }
    } 
}
std::vector<std::pair<int, int>> find_keypoint_matches(std::vector<Keypoint>& a,
                                                       std::vector<Keypoint>& b,
                                                       float thresh_relative,
                                                       float thresh_absolute)
{
    assert(a.size() >= 2 && b.size() >= 2);

    std::vector<std::pair<int, int>> matches;

    int *d_matches;
    int maxSizeMatches = (a.size() * 2 + 1);
    hipMalloc(&d_matches, maxSizeMatches * sizeof(int));
    hipMemset(d_matches, 0, maxSizeMatches * sizeof(int));

    uint8_t *d_a_descriptors;
    uint8_t *d_b_descriptors;
    int desc_size = 128;

    hipMalloc((void**)&d_a_descriptors, a.size() * desc_size * sizeof(uint8_t));
    hipMalloc((void**)&d_b_descriptors, b.size() * desc_size * sizeof(uint8_t));

    for(int i=0;i<a.size();i++)
        hipMemcpy(&d_a_descriptors[i*desc_size], &a[i].descriptor[0], desc_size * sizeof(uint8_t), hipMemcpyHostToDevice);
    
    for(int i=0;i<b.size();i++)
        hipMemcpy(&d_b_descriptors[i*desc_size], &b[i].descriptor[0], desc_size * sizeof(uint8_t), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (a.size() + threadsPerBlock - 1) / threadsPerBlock;
    find_matches<<<blocksPerGrid, threadsPerBlock>>>(d_a_descriptors, d_b_descriptors, d_matches, a.size(), b.size(), desc_size, thresh_relative, thresh_absolute);
    hipDeviceSynchronize();

    int hostMatches[maxSizeMatches];
    hipMemcpy(hostMatches, d_matches, maxSizeMatches*sizeof(int), hipMemcpyDeviceToHost);
    
    // hostMatches[0] holds the count 
    for(int i=1;i<hostMatches[0];i+=2)
        matches.emplace_back(hostMatches[i], hostMatches[i+1]);

    hipFree(d_matches);
    hipFree(d_a_descriptors);
    hipFree(d_b_descriptors);

    return matches;
}

Image draw_keypoints(const Image& img, const std::vector<Keypoint>& kps)
{
    Image res(img);
    if (img.channels == 1) {
        res = grayscale_to_rgb(res);
    }
    for (auto& kp : kps) {
        draw_point(res, kp.x, kp.y, 5);
    }
    return res;
}

Image draw_matches(const Image& a, const Image& b, std::vector<Keypoint>& kps_a,
                   std::vector<Keypoint>& kps_b, std::vector<std::pair<int, int>> matches)
{
    Image res(a.width+b.width, std::max(a.height, b.height), 3);

    for (int i = 0; i < a.width; i++) {
        for (int j = 0; j < a.height; j++) {
            res.set_pixel(i, j, 0, a.get_pixel(i, j, 0));
            res.set_pixel(i, j, 1, a.get_pixel(i, j, a.channels == 3 ? 1 : 0));
            res.set_pixel(i, j, 2, a.get_pixel(i, j, a.channels == 3 ? 2 : 0));
        }
    }
    for (int i = 0; i < b.width; i++) {
        for (int j = 0; j < b.height; j++) {
            res.set_pixel(a.width+i, j, 0, b.get_pixel(i, j, 0));
            res.set_pixel(a.width+i, j, 1, b.get_pixel(i, j, b.channels == 3 ? 1 : 0));
            res.set_pixel(a.width+i, j, 2, b.get_pixel(i, j, b.channels == 3 ? 2 : 0));
        }
    }

    for (auto& m : matches) {
        Keypoint& kp_a = kps_a[m.first];
        Keypoint& kp_b = kps_b[m.second];
        draw_line(res, kp_a.x, kp_a.y, a.width+kp_b.x, kp_b.y);
    }
    return res;
}

} // namespace parallel_cuda_sift
